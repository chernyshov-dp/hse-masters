#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1024
#define THREADS_PER_BLOCK 16

__global__ void matrix_multiplication(double *A, double *B, double *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        double sum = 0.0;
        for (int i = 0; i < n; i++) {
            sum += A[row * n + i] * B[i * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    double *hA, *hB, *hC; 
    double *dA, *dB, *dC;

    // Using Pinned Memory
    hipHostMalloc((void**)&hA, N * N * sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&hB, N * N * sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&hC, N * N * sizeof(double), hipHostMallocDefault);

    for (int i = 0; i < N * N; i++) {
        hA[i] = (double)rand() / RAND_MAX;
        hB[i] = (double)rand() / RAND_MAX;
    }

    hipMalloc((void**)&dA, N * N * sizeof(double));
    hipMalloc((void**)&dB, N * N * sizeof(double));
    hipMalloc((void**)&dC, N * N * sizeof(double));

    hipMemcpy(dA, hA, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, N * N * sizeof(double), hipMemcpyHostToDevice);

    dim3 gridDim((N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    dim3 blockDim(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

    matrix_multiplication<<<gridDim, blockDim>>>(dA, dB, dC, N);

    hipMemcpy(hC, dC, N * N * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", hC[i * N + j]);
        }
        printf("\n");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);

    return 0;
}
