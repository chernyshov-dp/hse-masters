#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1024
#define THREADS_PER_BLOCK 16

__global__ void matrix_multiplication(double *A, double *B, double *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        double sum = 0.0;
        for (int i = 0; i < n; i++) {
            sum += A[row * n + i] * B[i * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    double *hA, *hB, *hC;
    double *dA, *dB, *dC;

    int nStreams = 4;
    int size = N * N / nStreams;

    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Using Pinned Memory
    hipHostMalloc((void**)&hA, N * N * sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&hB, N * N * sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&hC, N * N * sizeof(double), hipHostMallocDefault);
    
    for (int i = 0; i < N * N; i++) {
        hA[i] = (double)rand() / RAND_MAX;
        hB[i] = (double)rand() / RAND_MAX;
    }
    
    hipMalloc((void**)&dA, N * N * sizeof(double));
    hipMalloc((void**)&dB, N * N * sizeof(double));
    hipMalloc((void**)&dC, N * N * sizeof(double));
    
    dim3 gridDim((N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    dim3 blockDim(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    
    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; i++) {
        hipStreamCreate(&stream[i]);
    }
    
    hipEventRecord(start, 0);
    
    for (int i = 0; i < nStreams; i++) {
        hipMemcpyAsync(dA + i * size, hA + i * size, sizeof(double) * size, hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(dB + i * size, hB + i * size, sizeof(double) * size, hipMemcpyHostToDevice, stream[i]);
        
        matrix_multiplication<<<gridDim, blockDim, 0, stream[i]>>>(dA + i * size, dB + i * size, dC + i * size, N);
        
        hipMemcpyAsync(hC + i * size, dC + i * size, sizeof(double) * size, hipMemcpyDeviceToHost, stream[i]);
    }
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    printf("Elapsed time: %.3f ms\n", elapsedTime);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    for (int i = 0; i < nStreams; i++) {
        hipStreamSynchronize(stream[i]);
        hipStreamDestroy(stream[i]);
    }

    for (int j = 0; j < N; j++) {
        for (int i = 0; i < N; i++) {
        printf("%f ", hC[j * N + i]);
        }
    printf("\n");
    }   
    
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    
    return 0;
}