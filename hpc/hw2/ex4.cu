#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define N 1024

int main() {
    double *hA, *hB, *hC;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMallocManaged(&hA, N * N * sizeof(double));
    hipMallocManaged(&hB, N * N * sizeof(double));
    hipMallocManaged(&hC, N * N * sizeof(double));

    for (int i = 0; i < N * N; i++) {
        hA[i] = (double)rand() / RAND_MAX;
        hB[i] = (double)rand() / RAND_MAX;
    }

    double alpha = 1.0;
    double beta = 0.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, hA, N, hB, N, &beta, hC, N);

    hipDeviceSynchronize();

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", hC[i * N + j]);
        }
        printf("\n");
    }

    hipFree(hA);
    hipFree(hB);
    hipFree(hC);

    hipblasDestroy(handle);

    return 0;
}
